/*
Copyright 2021 Fixstars Corporation
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http ://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include <cmath>


static constexpr int NUM_TRIALS = 5;


// CPU版行列積カーネル
void matmul_cpu(float *C, const float *A, const float *B, int n){
	for(int i = 0; i < n; ++i){
		for(int j = 0; j < n; ++j){
			C[i * n + j] = 0.0f;
			for(int k = 0; k < n; ++k){
				C[i * n + j] += A[i * n + k] * B[k * n + j];
			}
		}
	}
}

// CPU版処理時間計測
// NUM_TRIALS 回計測して中央値を求める
double matmul_cpu_benchmark(float *C, const float *A, const float *B, int n){
	std::vector<double> durations(NUM_TRIALS);
	for(int i = 0; i < NUM_TRIALS; ++i){
		const auto begin = std::chrono::steady_clock::now();
		matmul_cpu(C, A, B, n);
		const auto end = std::chrono::steady_clock::now();
		const auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - begin);
		durations[i] = duration.count() * 1e-3;
	}
	std::sort(durations.begin(), durations.end());
	return durations[NUM_TRIALS / 2];
}


// GPU版行列積カーネル
__global__ void matmul_gpu(float *C, const float *A, const float *B, int n){
	const int i = blockIdx.y * blockDim.y + threadIdx.y;
	const int j = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	for(int k = 0; k < n; ++k){
		sum += A[i * n + k] * B[k * n + j];
	}
	C[i * n + j] = sum;
}

void call_matmul_gpu(float *C, const float *A, const float *B, int n){
	const dim3 bdim(16, 16, 1), gdim(n / 16, n / 16, 1);
	matmul_gpu<<<gdim, bdim>>>(C, A, B, n);
}

// GPU版処理時間計測
// NUM_TRIALS 回計測して中央値を求める
double matmul_gpu_benchmark(float *h_C, const float *h_A, const float *h_B, int n){
	// デバイスメモリの確保
	float *d_C = nullptr, *d_A = nullptr, *d_B = nullptr;
	hipMalloc(&d_A, sizeof(float) * n * n);
	hipMalloc(&d_B, sizeof(float) * n * n);
	hipMalloc(&d_C, sizeof(float) * n * n);
	// 入力データの転送
	hipMemcpy(d_A, h_A, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(float) * n * n, hipMemcpyHostToDevice);

	std::vector<double> durations(NUM_TRIALS);
	for(int i = 0; i < NUM_TRIALS; ++i){
		const auto begin = std::chrono::steady_clock::now();
		call_matmul_gpu(d_C, d_A, d_B, n);
		hipDeviceSynchronize();  // GPUカーネルの終了を待つ
		const auto end = std::chrono::steady_clock::now();
		const auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - begin);
		durations[i] = duration.count() * 1e-3;
	}

	// 出力データの転送
	hipMemcpy(h_C, d_C, sizeof(float) * n * n, hipMemcpyDeviceToHost);
	// デバイスメモリの開放
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// 中央値を求める
	std::sort(durations.begin(), durations.end());
	return durations[NUM_TRIALS / 2];
}


// 検算
bool validate(const float *expect, const float *actual, int n){
	bool valid = true;
	for(int i = 0; i < n * n; ++i){
		if(std::fabs(expect[i] - actual[i]) > 1e-4){
			std::cerr << "(" << i / n << ", " << i % n << "): " << expect[i] << " != " << actual[i] << std::endl;
			valid = false;
		}
	}
	return valid;
}


int main(int argc, char *argv[]){
	if(argc < 2){
		std::cerr << "Usage: " << argv[0] << " n" << std::endl;
		return 0;
	}

	const int n = atoi(argv[1]);
	std::cout << "n = " << n << std::endl;

	std::default_random_engine engine;
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

	std::vector<float> A(n * n), B(n * n), cpu_C(n * n), gpu_C(n * n);
	for(int i = 0; i < n * n; ++i){
		A[i] = dist(engine);
		B[i] = dist(engine);
	}

	const auto cpu_duration =
		matmul_cpu_benchmark(cpu_C.data(), A.data(), B.data(), n);
	std::cout << "CPU: " << cpu_duration << " [ms]" << std::endl;

	const auto gpu_duration =
		matmul_gpu_benchmark(gpu_C.data(), A.data(), B.data(), n);
	std::cout << "GPU: " << gpu_duration << " [ms]" << std::endl;

	const auto valid = validate(cpu_C.data(), gpu_C.data(), n);
	std::cout << "Validation: " << (valid ? "Success" : "Failed") << std::endl;

	return 0;
}
